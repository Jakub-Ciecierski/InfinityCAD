#include "hip/hip_runtime.h"
//
// Created by jakub on 4/17/16.
//

#include <ifc_gpu/projection/ndc_gpu.h>
#include <iostream>

using namespace glm;

__global__
void ndc_kernel(vec4* d_vertices, int count, mat4* d_MVP){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < count){
        vec4& v = d_vertices[tid];

        v = (*d_MVP) * v;

        v.x /= v.w;
        v.y /= v.w;
        v.z /= v.w;
    }
}

__host__
void ndc_init(vec4* d_vertices, int count, mat4* d_MVP){
    dim3 DimGrid((count / MAX_BLOCK_THREAD_COUNT) + 1, 1, 1);
    dim3 DimBlock(MAX_BLOCK_THREAD_COUNT, 1, 1);

    ndc_kernel << <DimGrid, DimBlock>> >(d_vertices, count, d_MVP);

    hipDeviceSynchronize();
}

__host__
void ndc_compute(const vec4* vertices, int count,
                 const mat4* MVP, vec4** ndc_vertices){
    //std::cout << "ndc_compute" << std::endl;
    int verticesMemSize = count * sizeof(vec4);
    int mvpMemSize = sizeof(mat4);

    vec4* d_vertices;
    mat4* d_MVP;

    hipError_t err;

    if ((err = hipMalloc((void**)&d_vertices, verticesMemSize)) != hipSuccess)
        C_ERR(err);
    if ((err = hipMalloc((void**)&d_MVP, mvpMemSize)) != hipSuccess)
        C_ERR(err);

    if ((err = hipMemcpy(d_vertices, vertices, verticesMemSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);
    if ((err = hipMemcpy(d_MVP, MVP, mvpMemSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);

    ndc_init(d_vertices, count, d_MVP);

    if ((err = hipMemcpy(*ndc_vertices, d_vertices, verticesMemSize,
                          hipMemcpyDeviceToHost)) != hipSuccess) C_ERR(err);

    if ((err = hipFree(d_vertices)) != hipSuccess) C_ERR(err);
    if ((err = hipFree(d_MVP)) != hipSuccess) C_ERR(err);
}