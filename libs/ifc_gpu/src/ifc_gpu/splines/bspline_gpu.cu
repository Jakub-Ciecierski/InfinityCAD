#include "hip/hip_runtime.h"
//
// Created by jakub on 4/17/16.
//

#include <iostream>
#include "ifc_gpu/splines/bspline_gpu.h"

using namespace glm;
using namespace ifc_gpu;

__device__
float ifc_gpu::bsplineRecursiveGPU(float t, int n, int i,
                          float* knotVector){
    if (n == 0){
        if(t >= knotVector[i] && t < knotVector[i+1])
            return 1;
        else
            return 0;
    }
    float leftRecursion = (t - knotVector[i]);
    float leftDenominator = (knotVector[i+n] - knotVector[i]);
    if(leftRecursion == 0 || leftDenominator == 0)
        leftRecursion = 1;
    else
        leftRecursion /= leftDenominator;

    float rightRecursion = knotVector[i+1+n] - t;
    float rightDenominator = (knotVector[i+1+n] - knotVector[i+1]);
    if(rightRecursion == 0 || rightDenominator == 0)
        rightRecursion = 1;
    else
        rightRecursion /= rightDenominator;

    leftRecursion *= bsplineRecursiveGPU(t, n-1, i, knotVector);
    rightRecursion *= bsplineRecursiveGPU(t, n-1, i+1, knotVector);

    return leftRecursion + rightRecursion;
}

__global__
void ifc_gpu::computeBSplineKernel(vec3* points, int pointCount,
                                   float* knotVector,
                                   vec4* outCurve, int outSize,
                                   float tstart, float dt, int degree,
                                   mat4* MVP){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < outSize){
        float x,y,z;
        x = y = z = 0;
        float t = tstart + tid*dt;
        for(int i = 0;i < pointCount;i++){
            vec3& p = points[i];

            float bsplineBasisValue = bsplineRecursiveGPU(t, degree, i,
                                                          knotVector);
            x += p.x * bsplineBasisValue;
            y += p.y * bsplineBasisValue;
            z += p.z * bsplineBasisValue;
        }

        vec4 v = vec4(x,y,z,1);
        v = (*MVP) * v;

        v.x /= v.w;
        v.y /= v.w;

        outCurve[tid] = v;
    }
}

__host__
void ifc_gpu::computeBSplineInit(glm::vec3* d_points, int pointCount,
                        float* d_knotVector,
                        glm::vec4* d_outCurve, int outSize,
                        float tstart, float dt, int degree,
                        glm::mat4* MVP){
    dim3 DimGrid((outSize / MAX_BLOCK_THREAD_COUNT) + 1, 1, 1);
    dim3 DimBlock(MAX_BLOCK_THREAD_COUNT, 1, 1);

    computeBSplineKernel << <DimGrid, DimBlock>> >
                    (
                            d_points, pointCount, d_knotVector,
                            d_outCurve, outSize,
                            tstart, dt, degree, MVP);

    hipDeviceSynchronize();
}

__host__
void ifc_gpu::computeBSpline(vec3* points, int pointCount,
                    float* knotVector, int knotCount,
                    vec4* outCurve, int outSize,
                    float tstart, float dt, int degree,
                    const mat4* MVP){
    int pointsMemSize = pointCount * sizeof(vec3);
    int knotsMemSize = knotCount * sizeof(float);
    int outputMemSize = outSize * sizeof(vec4);
    int mvpMemSize = sizeof(mat4);

    vec3* d_points;
    float* d_knotVector;
    vec4* d_outCurve;
    mat4* d_MVP;

    hipError_t err;

    if ((err = hipMalloc((void**)&d_points, pointsMemSize)) != hipSuccess)
        C_ERR(err);
    if ((err = hipMalloc((void**)&d_knotVector, knotsMemSize)) != hipSuccess)
        C_ERR(err);
    if ((err = hipMalloc((void**)&d_outCurve, outputMemSize)) != hipSuccess)
        C_ERR(err);
    if ((err = hipMalloc((void**)&d_MVP, mvpMemSize)) != hipSuccess)
        C_ERR(err);

    if ((err = hipMemcpy(d_points, points, pointsMemSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);
    if ((err = hipMemcpy(d_knotVector, knotVector, knotsMemSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);
    if ((err = hipMemcpy(d_outCurve, outCurve, outSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);
    if ((err = hipMemcpy(d_MVP, MVP, mvpMemSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);

    computeBSplineInit(d_points, pointCount, d_knotVector,
                       d_outCurve, outSize, tstart, dt, degree, d_MVP);

    if ((err = hipMemcpy(outCurve, d_outCurve, outputMemSize,
                          hipMemcpyDeviceToHost)) != hipSuccess) C_ERR(err);

    if ((err = hipFree(d_points)) != hipSuccess) C_ERR(err);
    if ((err = hipFree(d_knotVector)) != hipSuccess) C_ERR(err);
    if ((err = hipFree(d_outCurve)) != hipSuccess) C_ERR(err);
    if ((err = hipFree(d_MVP)) != hipSuccess) C_ERR(err);
}