#include "hip/hip_runtime.h"
//
// Created by jakub on 5/3/16.
//

#include "ifc_gpu/surfaces/bezier_surface_gpu.h"

using namespace glm;

__global__
void ifc_gpu::computeBezierSurfaceKernel(glm::mat4* d_xComponents,
                                         glm::mat4* d_yComponents,
                                         glm::mat4* d_zComponents,
                                         int patchCount,
                                         float u_min, float u_max,
                                         float v_min,  float v_max,
                                         float du, float dv,
                                         glm::vec4* d_surfacePixels, int pixelCount,
                                         glm::mat4* d_MVP){
    int pixelsPerPatch = pixelCount / patchCount;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < pixelCount){
        int patchID = tid / pixelsPerPatch;
        int patchPixel = tid % pixelsPerPatch;

        float u;
        //float t = tstart + tid*dt;
    }
}

__host__
void ifc_gpu::computeBezierSurfaceInit(glm::mat4* d_xComponents,
                                       glm::mat4* d_yComponents,
                                       glm::mat4* d_zComponents,
                                       int patchCount,
                                       float u_min, float u_max,
                                       float v_min,  float v_max,
                                       float du, float dv,
                                       glm::vec4* d_surfacePixels,
                                       int pixelCount,
                                       glm::mat4* d_MVP){
    dim3 DimGrid((pixelCount / MAX_BLOCK_THREAD_COUNT) + 1, 1, 1);
    dim3 DimBlock(MAX_BLOCK_THREAD_COUNT, 1, 1);

    computeBezierSurfaceKernel << <DimGrid, DimBlock>> >
                                                (d_xComponents, d_yComponents,
                                                        d_zComponents,
                                                        patchCount,
                                                        u_min, u_max,
                                                        v_min, v_max,
                                                        du, dv,
                                                        d_surfacePixels,
                                                        pixelCount,
                                                        d_MVP);
    hipDeviceSynchronize();
}

void ifc_gpu::computeBezierSurface(const mat4* xComponents,
                                   const mat4* yComponents,
                                   const mat4* zComponents,
                                   int patchCount,
                                   float u_min, float u_max,
                                   float v_min, float v_max,
                                   float du, float dv,
                                   vec4* surfacePixels, int pixelCount,
                                   const mat4* MVP){
    int pointComponentsMemSize = patchCount*sizeof(mat4);
    int surfacePixelsMemSize = pixelCount * sizeof(vec4);
    int mvpMemSize = sizeof(mat4);

    mat4* d_xComponents;
    mat4* d_yComponents;
    mat4* d_zComponents;
    vec4* d_surfacePixels;
    mat4* d_MVP;

    hipError_t err;

    if ((err = hipMalloc((void**)&d_xComponents, pointComponentsMemSize)) != hipSuccess)
        C_ERR(err);
    if ((err = hipMalloc((void**)&d_yComponents, pointComponentsMemSize)) != hipSuccess)
        C_ERR(err);
    if ((err = hipMalloc((void**)&d_zComponents, pointComponentsMemSize)) != hipSuccess)
        C_ERR(err);
    if ((err = hipMalloc((void**)&d_surfacePixels, surfacePixelsMemSize)) != hipSuccess)
        C_ERR(err);
    if ((err = hipMalloc((void**)&mvpMemSize, mvpMemSize)) != hipSuccess)
        C_ERR(err);


    if ((err = hipMemcpy(d_xComponents, xComponents, pointComponentsMemSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);
    if ((err = hipMemcpy(d_yComponents, yComponents, pointComponentsMemSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);
    if ((err = hipMemcpy(d_zComponents, zComponents, pointComponentsMemSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);
    if ((err = hipMemcpy(d_MVP, MVP, mvpMemSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);

    // go

    computeBezierSurfaceInit(d_xComponents, d_yComponents, d_zComponents,
    patchCount, u_min, u_max, v_min, v_max, du, dv, d_surfacePixels,
                             pixelCount, d_MVP);

    if ((err = hipMemcpy(d_surfacePixels, surfacePixels, surfacePixelsMemSize,
                          hipMemcpyDeviceToHost)) != hipSuccess) C_ERR(err);

    if ((err = hipFree(d_xComponents)) != hipSuccess) C_ERR(err);
    if ((err = hipFree(d_yComponents)) != hipSuccess) C_ERR(err);
    if ((err = hipFree(d_zComponents)) != hipSuccess) C_ERR(err);
    if ((err = hipFree(d_surfacePixels)) != hipSuccess) C_ERR(err);
    if ((err = hipFree(d_MVP)) != hipSuccess) C_ERR(err);
}
