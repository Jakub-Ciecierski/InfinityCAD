#include "hip/hip_runtime.h"
//
// Created by jakub on 5/3/16.
//

#include <iostream>
#include "ifc_gpu/surfaces/bezier_surface_gpu.h"

using namespace glm;

__device__
float ifc_gpu::getMultplicationValueGPU(const vec4& v1, const mat4& m, const vec4& v2){
    vec4 res = v1 * m * v2;
    float val = res.x + res.y + res.z + res.w;

    return val;
}

__device__
vec4 ifc_gpu::cubicBernsteinVectorGPU(float t){
    float t2 = t*t;
    float t3 = t*t*t;

    float B0 = 1 - 3*t + 3*t2 - t3;
    float B1 = 3*t - 6*t2 + 3*t3;
    float B2 = 3*t2 - 3*t3;
    float B3 = t3;

    return vec4(B0, B1, B2, B3);
}

__global__
void ifc_gpu::computeBezierSurfaceKernel(glm::mat4* d_xComponents,
                                         glm::mat4* d_yComponents,
                                         glm::mat4* d_zComponents,
                                         int patchCount,
                                         glm::vec2* uvPatchParameters,
                                         int paramCount,
                                         glm::vec4* d_surfacePixels, int pixelCount,
                                         glm::mat4* d_MVP){
    int pixelsPerPatch = pixelCount / patchCount;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < pixelCount){
        int patchID = tid / pixelsPerPatch;
        if(patchID < patchCount){
            int patchPixel = tid % pixelsPerPatch;

            vec2 params = uvPatchParameters[patchPixel];

            float u =  params.x;
            float v =  params.y;

            vec4 Bu = cubicBernsteinVectorGPU(u);
            vec4 Bv = cubicBernsteinVectorGPU(v);

            float x = getMultplicationValueGPU(Bu, d_xComponents[patchID], Bv);
            float y = getMultplicationValueGPU(Bu, d_yComponents[patchID], Bv);
            float z = getMultplicationValueGPU(Bu, d_zComponents[patchID], Bv);

            vec4 point(x, y, z, 1);
            point = *d_MVP * point;

            point.x /= point.w;
            point.y /= point.w;

            d_surfacePixels[tid] = point;
        }

    }
}

__host__
void ifc_gpu::computeBezierSurfaceInit(glm::mat4* d_xComponents,
                                       glm::mat4* d_yComponents,
                                       glm::mat4* d_zComponents,
                                       int patchCount,
                                       glm::vec2* uvPatchParameters,
                                       int paramCount,
                                       glm::vec4* d_surfacePixels,
                                       int pixelCount,
                                       glm::mat4* d_MVP){
    dim3 DimGrid((pixelCount / MAX_BLOCK_THREAD_COUNT) + 1, 1, 1);
    dim3 DimBlock(MAX_BLOCK_THREAD_COUNT, 1, 1);

    computeBezierSurfaceKernel << <DimGrid, DimBlock>> >
                                                (d_xComponents, d_yComponents,
                                                        d_zComponents,
                                                        patchCount,
                                                        uvPatchParameters,
                                                        paramCount,
                                                        d_surfacePixels,
                                                        pixelCount,
                                                        d_MVP);
    hipDeviceSynchronize();
}

__host__
void ifc_gpu::computeBezierSurface(mat4* xComponents,
                                   mat4* yComponents,
                                   mat4* zComponents,
                                   int patchCount,
                                   glm::vec2* uvPatchParameters,
                                   int paramCount,
                                   vec4* surfacePixels, int pixelCount,
                                   const mat4* MVP){
    size_t pointComponentsMemSize = patchCount*sizeof(mat4);
    size_t surfacePixelsMemSize = pixelCount * sizeof(vec4);
    size_t uvPatchParametersMemSize = paramCount * sizeof(vec2);
    size_t mvpMemSize = sizeof(mat4);

    mat4* d_xComponents;
    mat4* d_yComponents;
    mat4* d_zComponents;
    vec2* d_uvPatchParameters;
    vec4* d_surfacePixels;
    mat4* d_MVP;

    hipError_t err;

    if ((err = hipMalloc((void**)&d_MVP, mvpMemSize)) != hipSuccess)
        C_ERR(err);

    if ((err = hipMalloc((void**)&d_xComponents, pointComponentsMemSize)) != hipSuccess)
        C_ERR(err);

    if ((err = hipMalloc((void**)&d_yComponents, pointComponentsMemSize)) != hipSuccess)
        C_ERR(err);

    if ((err = hipMalloc((void**)&d_zComponents, pointComponentsMemSize)) != hipSuccess)
        C_ERR(err);

    if ((err = hipMalloc((void**)&d_uvPatchParameters, uvPatchParametersMemSize)) != hipSuccess)
        C_ERR(err);

    if ((err = hipMalloc((void**)&d_surfacePixels, surfacePixelsMemSize)) != hipSuccess)
        C_ERR(err);


    if ((err = hipMemcpy(d_xComponents, xComponents, pointComponentsMemSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);
    if ((err = hipMemcpy(d_yComponents, yComponents, pointComponentsMemSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);
    if ((err = hipMemcpy(d_zComponents, zComponents, pointComponentsMemSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);
    if ((err = hipMemcpy(d_uvPatchParameters, uvPatchParameters, uvPatchParametersMemSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);
    if ((err = hipMemcpy(d_MVP, MVP, mvpMemSize,
                          hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);


    computeBezierSurfaceInit(d_xComponents, d_yComponents, d_zComponents,
                             patchCount,
                             d_uvPatchParameters, paramCount,
                             d_surfacePixels,
                             pixelCount, d_MVP);

    if ((err = hipMemcpy(surfacePixels, d_surfacePixels, surfacePixelsMemSize,
                          hipMemcpyDeviceToHost)) != hipSuccess) C_ERR(err);

    if ((err = hipFree(d_xComponents)) != hipSuccess) C_ERR(err);
    if ((err = hipFree(d_yComponents)) != hipSuccess) C_ERR(err);
    if ((err = hipFree(d_zComponents)) != hipSuccess) C_ERR(err);
    if ((err = hipFree(d_uvPatchParameters)) != hipSuccess) C_ERR(err);
    if ((err = hipFree(d_surfacePixels)) != hipSuccess) C_ERR(err);
    if ((err = hipFree(d_MVP)) != hipSuccess) C_ERR(err);
}